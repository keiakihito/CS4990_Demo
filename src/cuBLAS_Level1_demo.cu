#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../include/checks.h"
#include "../include/helper.h"

int main(){
    const int N = 5;
    const float vecX_h[N] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    const float vecY_h[N] = {5.0f, 4.0f, 3.0f, 2.0f, 1.0f};
    float result_h = 0.0f;

    float* vecX_d = NULL;
    float* vecY_d = NULL;
    float* result_d = NULL;

    bool debug = true;

    // Allocate memory on device
    CHECK(hipMalloc((void**)&vecX_d, N * sizeof(float)));
    CHECK(hipMalloc((void**)&vecY_d, N * sizeof(float)));
    CHECK(hipMalloc((void**)&result_d, sizeof(float)));

    // Copy data from host to device
    CHECK(hipMemcpy(vecX_d, vecX_h, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(vecY_d, vecY_h, N * sizeof(float), hipMemcpyHostToDevice));

    if(debug){
        printf("\nVector X (Device Memory):\n");
        print_vector_d(vecX_d, N);

        printf("\nVector Y (Device Memory):\n");
        print_vector_d(vecY_d, N);

    }

    // Setup cuBLAS handler
    hipblasHandle_t cublasHandler = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasHandler));

    // Calling cuBLAS API
    CHECK_CUBLAS(hipblasSdot(cublasHandler, N, vecX_d, 1, vecY_d, 1, result_d));

    // Copy data from device to host
    CHECK(hipMemcpy(&result_h, result_d, sizeof(float), hipMemcpyDeviceToHost));

    //Check the result
    printf("\nresult_h: %f",result_h);
    // Free memeory
    CHECK_CUBLAS(hipblasDestroy(cublasHandler));
    CHECK(hipFree(vecX_d));
    CHECK(hipFree(vecY_d));
    CHECK(hipFree(result_d));

    return 0;
}

/*
Sample Run

Vector X (Device Memory):
1.0000000000
2.0000000000
3.0000000000
4.0000000000
5.0000000000

Vector Y (Device Memory):
5.0000000000
4.0000000000
3.0000000000
2.0000000000
1.0000000000

result_h: 35.000000

*/