#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "../include/checks.h"  
#include "../include/helper.h" 

int main(){
    // Marix dimensions
    const int M = 3; // Rows of Matrix A and Matrix C
    const int N = 3; // Columns of Matrix B and Matrix C
    const int K = 3; // Columns of Matrix A and Rows of Matrx B

    const float alpha = 1.0f;
    const float beta = 0.0f;

    bool debug = true;

    // Row-major mattrices A, B and C
    float mtxA_h[M * K] = {
        1.0f, 2.0f, 3.0f,
        4.0f, 5.0f, 6.0f,
        7.0f, 8.0f, 9.0f
    };

    float mtxB_h[K * N] = {
        9.0f, 8.0f, 7.0f,
        6.0f, 5.0f, 4.0f,
        3.0f, 2.0f, 1.0f
    };

    float mtxC_h[M * N] = {
        0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f
    };

    float* mtxA_d = NULL;
    float* mtxB_d = NULL;
    float* mtxC_d = NULL;

    // Allocate memeory on the device
    CHECK(hipMalloc((void**)&mtxA_d, M * K * sizeof(float)));
    CHECK(hipMalloc((void**)&mtxB_d, K * N * sizeof(float)));
    CHECK(hipMalloc((void**)&mtxC_d, M * N * sizeof(float)));

    // Copy data from host to device
    CHECK(hipMemcpy(mtxA_d, mtxA_h, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(mtxB_d, mtxB_h, K * N * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(mtxC_d, mtxC_h, M * N * sizeof(float), hipMemcpyHostToDevice));

    if(debug){
        // Debug: Print device memory before operation
        printf("\nMatrix A (Device Memory, Row-Major):\n");
        print_mtx_row_d(mtxA_d, M, K);

        printf("\nMatrix B (Device Memory, Row-Major):\n");
        print_mtx_row_d(mtxB_d, K, N);

        printf("\nMatrix C (Device Memory, Row-Major):\n");
        print_mtx_row_d(mtxC_d, M, N);
    }

    // Set up cuBLAS
    hipblasHandle_t cublasHandler = NULL;
    CHECK_CUBLAS(hipblasCreate(&cublasHandler));

    // Call cuBLAS API
    // Since matrices are Row-Major, the leading dimensnion is as follow
    // Matrix A: K, Matrix B: N, Matrix C: K
    // cuBLAS assumes column-major order, so we need to transpose when it handles Row-major matrics
    CHECK_CUBLAS(hipblasSgemm(cublasHandler, HIPBLAS_OP_T,HIPBLAS_OP_T, M, N, K, &alpha, mtxA_d, K, mtxB_d, N, &beta, mtxC_d, K));

    if (debug){
        // Debug: Print device memory after operation
        printf("\nMatrix C (Device Memory, After SGEMM):\n");
        print_mtx_row_d(mtxC_d, M, N);
    }

    // Copy result from device to host
    CHECK(hipMemcpy(mtxC_h, mtxC_d, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Print the result Matrix C in host memory
    printf("\nMatrix C (Host Memory): \n");
    print_mtx_row_h(mtxC_h, M, N);

    // Free device memory
    CHECK_CUBLAS(hipblasDestroy(cublasHandler));
    CHECK(hipFree(mtxA_d));
    CHECK(hipFree(mtxB_d));
    CHECK(hipFree(mtxC_d));





    return 0;
}
