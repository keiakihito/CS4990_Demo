#include <iostream>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

#include "../include/checks.h"  
#include "../include/helper.h" 

int main(){
    // Matrix dimension
    const int numOfRow = 3;
    const int numOfClm = 3;
    const int nnz = 4; // Number of non-zero value

    const float alpha = 1.0f;
    const float beta = 0.0f;

  
    /*
    CSR format of Matrix A (Row-Major)
    Let Sparse Matrix A = 
    |1.0 0.0 0.0|
    |0.0 2.0 3.0|
    |0.0 0.0 4.0|
    */
    int csrOffsets_h[numOfRow+1] = {0, 1, 3, 4};
    int columns_h[nnz] = {0, 1, 2, 2};
    float values_h[nnz] = {1.0f, 2.0f, 3.0f, 4.0f};

    // Dense vector x and y
    float vecX_h[] = {1.0f, 2.0f, 3.0f};
    float vecY_h[] = {0.0f, 0.0f, 0.0f}; // Store result later

    // Device memeory assignments
    int *csrOffsets_d = NULL;
    int *columns_d = NULL;
    float *values_d = NULL;
    float *vecX_d = NULL;
    float *vecY_d = NULL;

    bool debug = true;

    // Allocate device memory
    CHECK(hipMalloc((void**)&csrOffsets_d, (numOfRow+1) * sizeof(int)));
    CHECK(hipMalloc((void**)&columns_d, nnz * sizeof(int)));
    CHECK(hipMalloc((void**)&values_d, nnz * sizeof(float)));
    CHECK(hipMalloc((void**)&vecX_d, numOfClm * sizeof(float)));
    CHECK(hipMalloc((void**)&vecY_d, numOfClm * sizeof(float)));

    // Copy data from host to device
    CHECK(hipMemcpy(csrOffsets_d, csrOffsets_h, (numOfRow + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(columns_d, columns_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(values_d, values_h, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(vecX_d, vecX_h, numOfClm * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(vecY_d, vecY_h, numOfRow * sizeof(float), hipMemcpyHostToDevice));

    if(debug){
        printf("\ncsrOffsets_d:\n");
        print_vector_d(csrOffsets_d, numOfRow+1);

        printf("\ncolumns_d:\n");
        print_vector_d(columns_d, nnz);

        printf("\nvaluse_d:\n");
        print_vector_d(values_d, nnz);

        printf("\nvecX_d: \n");
        print_vector_d(vecX_d, numOfRow);
        
        printf("\nvecY_d: \n");
        print_vector_d(vecY_d, numOfRow);
    }

    // Create cuSPARSE handler
    hipsparseHandle_t cusparseHandler = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandler));

    // Create matrix and vector descriptors
    hipsparseSpMatDescr_t mtxA_des = NULL;
    hipsparseDnVecDescr_t vecX_des = NULL;
    hipsparseDnVecDescr_t vecY_des = NULL;

    // Define sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&mtxA_des, numOfRow, numOfClm, nnz, csrOffsets_d, columns_d, values_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    // Define dense vecotrs x and y
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX_des, numOfRow, vecX_d, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY_des, numOfRow, vecY_d, HIP_R_32F));

    // Allocate workspase for SpMV
    size_t bufferSize = 0;
    void *buffer_d = NULL;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparseHandler, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,mtxA_des, vecX_des, &beta, vecY_des, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK(hipMalloc(&buffer_d, bufferSize));

    if(debug){
        printf("\nBuffer size (bytes): %zu\n", bufferSize);
    }

    // Perform SpMV
    CHECK_CUSPARSE(hipsparseSpMV(cusparseHandler, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, mtxA_des, vecX_des, &beta, vecY_des, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer_d));

    // Copy refult from device to host
    CHECK(hipMemcpy(vecY_h, vecY_d, numOfRow * sizeof(float), hipMemcpyDeviceToHost));

    printf("\nResult Vector vecY_h:\n");
    print_mtx_clm_h(vecY_h, numOfRow, 1);

    // Free memory
    CHECK_CUSPARSE(hipsparseDestroySpMat(mtxA_des));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX_des));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY_des));
    CHECK_CUSPARSE(hipsparseDestroy(cusparseHandler));
    CHECK(hipFree(csrOffsets_d));
    CHECK(hipFree(columns_d));
    CHECK(hipFree(values_d));
    CHECK(hipFree(vecX_d));
    CHECK(hipFree(vecY_d));
    CHECK(hipFree(buffer_d));

    return 0;
}

/*
Sample Run

csrOffsets_d:
0
1
3
4

columns_d:
0
1
2
2

valuse_d:
1.0000000000
2.0000000000
3.0000000000
4.0000000000

vecX_d: 
1.0000000000
2.0000000000
3.0000000000

vecY_d: 
0.0000000000
0.0000000000
0.0000000000

Buffer size (bytes): 8

Result Vector vecY_h:
1.000000 
13.000000 
12.000000 


 */