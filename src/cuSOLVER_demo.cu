/*
Reference page

Documentation
https://docs.nvidia.com/cuda/cusolver/index.html

Example code
https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/getrf/cusolver_getrf_example.cu

*/

#include <iostream>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

#include "../include/checks.h"
#include "../include/helper.h"

int main() {
    const int N = 2; // Matrix dimension (NxN)
    const int lda = N; // Leading dimension of A (since row-major, lda = N)

    /*
    Let matrix A
    |4.0 1.0|
    |3.0 1.0| 
    */ 

    // (1) Input matrix (Column-major order)
    float mtxA_h[lda * N] = {
        4.0f, 3.0f, 
        1.0f, 1.0f
    };


    bool debug = true;

    // (2) Device memory for matrix A
    float* mtxA_d = NULL;
    CHECK(hipMalloc((void**)&mtxA_d, lda * N * sizeof(float)));
    CHECK(hipMemcpy(mtxA_d, mtxA_h, lda * N * sizeof(float), hipMemcpyHostToDevice));

    if(debug){
        printf("\nmtxA_d before hipsolverDnSgetrf\n");
        print_mtx_clm_d(mtxA_d, N, N);
    }
    
    // (4) Pivot indices and info
    int* pivot_d = NULL;
    int* info_d = NULL;
    CHECK(hipMalloc((void**)&pivot_d, N * sizeof(int)));
    CHECK(hipMalloc((void**)&info_d, sizeof(int)));

    // (5) Set up cuSOLVER handle
    hipsolverHandle_t cusolverHandler = NULL;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandler));

    // (6) Workspace size and allocation
    int lwork = 0;
    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(cusolverHandler, N, N, mtxA_d, lda, &lwork));

    float* work_d = NULL;
    CHECK(hipMalloc((void**)&work_d, lwork * sizeof(float)));

    // (7) Perform LU factorization
    CHECK_CUSOLVER(hipsolverDnSgetrf(cusolverHandler, N, N, mtxA_d, lda, work_d, pivot_d, info_d));


    if(debug){
        // (8) Check the LU decompostion success or not.
        int info_h = 0; // Host-side info variable
        CHECK(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));

        // Check the info value
        if (info_h == 0) {
            printf("\nLU factorization successful.\n");
        } else if (info_h > 0) {
            printf("\nLU factorization failed. Matrix is singular at U[%d][%d].\n", info_h, info_h);
        } else if (info_h < 0) {
            printf("\nLU factorization failed. Illegal argument at position %d.\n", -info_h);
        }

        printf("\nmtxA_d after hipsolverDnSgetrf\n");
        print_mtx_clm_d(mtxA_d, N, N);
    }

    // (9) Feel free to explore
    // 1. solve AX = I to get inverse with hipsolverDnSgetrs
    // 2. Extract L and U explicitly 

    // (10) Clean up
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandler));
    CHECK(hipFree(mtxA_d));
    CHECK(hipFree(pivot_d));
    CHECK(hipFree(info_d));
    CHECK(hipFree(work_d));
    
    return 0;
}

/*
Sample Run

mtxA_d before hipsolverDnSgetrf
4.000000 1.000000 
3.000000 1.000000 

LU factorization successful.

mtxA_d after hipsolverDnSgetrf
4.000000 1.000000 
0.750000 0.250000 


*/