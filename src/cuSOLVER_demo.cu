/*
Reference page

Documentation
https://docs.nvidia.com/cuda/cusolver/index.html

Example code
https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/getrf/cusolver_getrf_example.cu

*/

#include <iostream>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

#include "../include/checks.h"
#include "../include/helper.h"

int main() {
    const int N = 2; // Matrix dimension (NxN)
    const int lda = N; // Leading dimension of A (since row-major, lda = N)

    /*
    Let matrix A
    |4.0 1.0|
    |3.0 1.0| 
    */ 

    // Input matrix (Column-major order)
    float mtxA_h[lda * N] = {
        4.0f, 3.0f, 
        1.0f, 1.0f
    };


    bool debug = true;

    // Device memory for matrix A
    float* mtxA_d = NULL;
    CHECK(hipMalloc((void**)&mtxA_d, lda * N * sizeof(float)));
    CHECK(hipMemcpy(mtxA_d, mtxA_h, lda * N * sizeof(float), hipMemcpyHostToDevice));

    if(debug){
        printf("\nmtxA_d before hipsolverDnSgetrf\n");
        print_mtx_clm_d(mtxA_d, N, N);
    }
    // Pivot indices and info
    int* pivot_d = NULL;
    int* info_d = NULL;
    CHECK(hipMalloc((void**)&pivot_d, N * sizeof(int)));
    CHECK(hipMalloc((void**)&info_d, sizeof(int)));

    // cuSOLVER handle
    hipsolverHandle_t cusolverHandler = NULL;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverHandler));

    // Workspace size and allocation
    int lwork = 0;
    CHECK_CUSOLVER(hipsolverDnSgetrf_bufferSize(cusolverHandler, N, N, mtxA_d, lda, &lwork));

    float* work_d = NULL;
    CHECK(hipMalloc((void**)&work_d, lwork * sizeof(float)));

    // Perform LU factorization
    CHECK_CUSOLVER(hipsolverDnSgetrf(cusolverHandler, N, N, mtxA_d, lda, work_d, pivot_d, info_d));

    if(debug){
        printf("\nmtxA_d after hipsolverDnSgetrf\n");
        print_mtx_clm_d(mtxA_d, N, N);
    }

    // Feel free to explore
    // 1. solve AX = I to get inverse with hipsolverDnSgetrs
    // 2. Extract L and U explicitly 

    // Clean up
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverHandler));
    CHECK(hipFree(mtxA_d));
    CHECK(hipFree(pivot_d));
    CHECK(hipFree(info_d));
    CHECK(hipFree(work_d));
    
    return 0;
}

/*
Sample Run

mtxA_d before hipsolverDnSgetrf
4.000000 1.000000 
3.000000 1.000000 

mtxA_d after hipsolverDnSgetrf
4.000000 1.000000 
0.750000 0.250000 
*/